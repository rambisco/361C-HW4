
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>

#define NUM_THREADS 512
#define NUM_BLOCKS 1

int* fileToArray(char file1[], int* n){
    FILE* fptr = fopen(file1, "r");
    char* str = (char*) malloc(sizeof(char)*2048);
    int token;
    fscanf(fptr, "%d,", n);
    int* array;
    hipMallocManaged(&array, sizeof(int)*(*n)); 

    for(int i = 0; i < *n; i++){
        fscanf(fptr, "%d,", &token);
        array[i] = token;
    }

    fclose(fptr);
    return array;
}

// we want to keep track of how many elements have a 0 in the current bit that is to be masked.
__global__ 
void sort(int* result, int* array, int mask, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n) {

    // this is the relative index
    result[index] = !(array[index] & mask);
  } 
}

__global__ 
void prescan(int* result, int* indices, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  //extern __shared__ int local_scan[];
  int from = blockIdx.x * blockDim.x;
  int to = blockIdx.x * blockDim.x + blockDim.x; 
   
  for (int d = 1; d < blockDim.x; d *= 2) {
    if (index + 1 - from > d) {
      indices[index] += indices[index-d];
    }
    __syncthreads();
  }
  result[index] = indices[index];
}

__global__
void map(int* result, int from) {
  int index = from + threadIdx.x;
  int to_map = result[from-1];
  result[index] += to_map;
  return;
}

__global__
void copy(int* result, int* indices, int* array, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n) {
      if (array[index] % 2 == 1) {
        int idx = indices[index];
        result[idx] = array[index];
      }
  }
}

void radixSort(int* array, int n) {
    int threads = 1024;
    int blocks = (n + (threads-1)) / threads;
    
    int* indices;  // stores whether each element in array is odd or not (denoted 1 or 0)
    int* prefix;  // stores prefix sum of each element
    int* result;  // stores final result, sizeof prefix[n-1]
    int local_array_bytes = sizeof(int)*threads;
  
    hipMallocManaged(&indices, sizeof(int) * n);
    hipMallocManaged(&prefix, sizeof(int) * n);
    hipMallocManaged(&result, sizeof(int) * n);

    for (int i = 1; i <= 1024; i <<= 2) {
        sort<<<blocks, threads>>>(indices, array, i, n);

        hipDeviceSynchronize();
        printf("indices[999999]: %d\n", indices[999999]);

        prescan<<<blocks, threads, local_array_bytes>>>(prefix, indices, n); 

        // so far we've only calculated the positions of elements with 0 in the bit of interest
        // we need to use new index of the last element with a 0 in the bit of interest as an offset 
        // to calculate the positions of elements with 1 in the bit of interest
        // also I think we need to copy results -> input array at the end of each iteration for each bit of interest
  
        hipDeviceSynchronize();
  
        for(int i = threads; i < n; i+=threads) {
        map<<<1, threads>>>(prefix, i); //map last value of previous group of 1024 onto next group of 1024
        hipDeviceSynchronize();
        }
    }
  
    printf("prescan[999998]: %d, prescan[999999]: %d\n", prefix[999998], prefix[999999]);
  
    int maxOdds = prefix[n] + 1;
    printf("max number of odds: %d\n", prefix[n]);
  
    copy<<<blocks, threads>>>(result, prefix, array, n);
  
    hipDeviceSynchronize();
  
     for(int i = maxOdds - 10; i < maxOdds; i++) {
       printf("index: %d result: %d\n", i, result[i]);
     }
  
  }


int main(int argc, char* argv[]){
    int n;
    int* array = fileToArray("inp.txt", &n);
    radixSort(array, n);
    hipFree(array);
  }