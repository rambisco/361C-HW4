
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>

#define NUM_THREADS 512
#define OUTPUT_FILE_NAME "q3.txt"
#define NUM_BLOCKS 1

// int* fileToArray(char file1[], int* n){
//     FILE* fptr = fopen(file1, "r");
//     FILE* fptr_cpy = fptr;
//     char* str = (char*) malloc(sizeof(char)*2048);
//     int token;
//     int count = 0;
//     while (fscanf(fptr, "%d ,", &token) != EOF) {
//       count++;
//     }
//     count++;
//     *n = count;
//     //fscanf(fptr, "%d,", n);
//     int* array;
//     cudaMallocManaged(&array, sizeof(int)*(*n));
//     int i = 0 
//     while (fscanf(fptr, "%d ,", &token) != EOF) {
//       array[i] = token;
//       i++;
//     }
//     i++;
//     fscanf(fptr, "%d", &token);
//     array[i] = token;
//     // for(int i = 0; i < *n; i++){
//     //     fscanf(fptr, "%d,", &token);
//     //     array[i] = token;
//     // }

//     fclose(fptr);
//     return array;
// }

int* fileToArray(char file1[], int* n){
  FILE* fptr = fopen(file1, "r");
  // char* str = (char*) malloc(sizeof(char)*2048);
  int token;
  int count = 0;
  while (fscanf(fptr, "%d, ", &token) != EOF) {
    //printf("%dth token: %d\n", count, token);
    count++;
  }
  *n = count;
  //printf("total number of elements: %d\n", *n);
  int* array;
  hipMallocManaged(&array, sizeof(int)*(*n));
  rewind(fptr);
  for(int i = 0; i < *n; i++){
      fscanf(fptr, "%d, ", &token);
      array[i] = token;
  }

  fclose(fptr);
  return array;
}

// we want to keep track of how many elements have a 0 in the current bit that is to be masked.
__global__ 
void maskArray(int* result2, int* result, int* array, int mask, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n) {

    // this is the relative index
    result[index] = (array[index] & mask) == mask ? 1 : 0;
    result2[index] = (array[index] & mask) == mask ? 0 : 1; 
  } 
}

__global__ 
void prescan(int* indices, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  //extern __shared__ int local_scan[];
  int from = blockIdx.x * blockDim.x;
  int to = blockIdx.x * blockDim.x + blockDim.x; 
   
  for (int d = 1; d < blockDim.x; d *= 2) {
    if (index + 1 - from > d && index < n) {
      indices[index] += indices[index-d];
    }
    __syncthreads();
  }
}

__global__
void map(int* result, int from) {
  int index = from + threadIdx.x;
  int to_map = result[from-1];
  result[index] += to_map;
  return;
}

__global__
void copy(int* result, int* array, int* ones, int* zeroes, int n, int pivot, int mask) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n) {
      if (array[index] & mask != 0) {
        int idx = ones[index];
        result[idx + pivot] = array[index];
      }
      else {
        int idx = zeroes[index];
        result[idx] = array[index];
      }
  }
}

void radixSort(int* array, int n) {
    int threads = 1024;
    int blocks = (n + (threads-1)) / threads;
    
    // stores whether each element in array is odd or not (denoted 1 or 0)
    int* ones;
    int* zeroes;
    int* result_copy;  // stores prefix sum of each element
    int* result;  // stores final result, sizeof prefix[n-1]
    int local_array_bytes = sizeof(int)*threads;

    hipMallocManaged(&result_copy, sizeof(int) * n);
    hipMallocManaged(&result, sizeof(int) * n);
    hipMallocManaged(&ones, sizeof(int) * n);
    hipMallocManaged(&zeroes, sizeof(int) * n);

    for (unsigned int i = 1; i <= 1024; i <<= 1) {

        printf("\n\nMask: %x\n\n", i); 

        maskArray<<<blocks, threads>>>(zeroes, ones, array, i, n);

        hipDeviceSynchronize();

        for(int j = 0; j < 10; j++){
          printf("zeroes[%d]: %d - ", n - j, zeroes[n - j]);  
          printf("ones[%d]: %d\n", n - j, ones[n - j]);
        }

        printf("zeroes[%d]: %d", 999999, zeroes[999999]);
        printf("ones[%d]: %d", 999999, ones[999999]);

        prescan<<<blocks, threads, local_array_bytes>>>(zeroes, n); 

        hipDeviceSynchronize();

        for(int j = threads; j < n; j+=threads) {
        map<<<1, threads>>>(zeroes, j); //map last value of previous group of 1024 onto next group of 1024
        hipDeviceSynchronize();
        }

        for(int j = 0; j < 10; j++){
          printf("zeroes[%d]: %d - ", j, zeroes[j]);  
          printf("zeroes[%d]: %d\n", threads - 2 + j, zeroes[threads - 2 + j]);
        }

        prescan<<<blocks, threads, local_array_bytes>>>(ones, n); 

        hipDeviceSynchronize();

        for(int j = threads; j < n; j+=threads) {
        map<<<1, threads>>>(ones, j); //map last value of previous group of 1024 onto next group of 1024
        hipDeviceSynchronize();
        }

        for(int j = 0; j < 10; j++){
          printf("ones[%d]: %d - ", j, ones[j]);  
          printf("ones[%d]: %d\n", threads - 2 + j, ones[threads - 2 + j]);
        }

        printf("zeroes[%d]: %d", 999999, zeroes[999999]);
        printf("ones[%d]: %d", 999999, ones[999999]);

        int pivot = zeroes[n-1];

        // so far we've only calculated the positions of elements with 0 in the bit of interest
        // we need to use new index of the last element with a 0 in the bit of interest as an offset 
        // to calculate the positions of elements with 1 in the bit of interest
        // also I think we need to copy results -> input array at the end of each iteration for each bit of interest
  
        hipDeviceSynchronize();

        copy<<<blocks, threads>>>(result, array, ones, zeroes, n, pivot, i);

        hipDeviceSynchronize();

        for(int j = 0; j < 10; j++) {
          printf("result[%d]: %d\n", j, result[j]);
        }
    }
    FILE *output = fopen(OUTPUT_FILE_NAME, "w");
    if(output == NULL) printf("failed to open file %s\n", OUTPUT_FILE_NAME);
    fprintf(output, "%d", result[0]);
    for(int i = 1; i < n ; i++) {
      fprintf(output, ", %d", result[i]);
    }
    fclose(output);
  }


int main(int argc, char* argv[]){
    int n;
    int* array = fileToArray("inp.txt", &n);
    printf("Number of elements in array: %d\n", n);
    for (int i = 0; i < n; i++) {
      printf("%d, ", array[i]);
    }
    radixSort(array, n);

    

    hipFree(array);
  }
