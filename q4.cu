#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>

#define NUM_THREADS 512
#define NUM_BLOCKS 1

int* fileToArray(char file1[], int* n){
    FILE* fptr = fopen(file1, "r");
    char* str = (char*) malloc(sizeof(char)*2048);
    int token;
    fscanf(fptr, "%d,", n);
    int* array;
    hipMallocManaged(&array, sizeof(int)*(*n)); 

    for(int i = 0; i < *n; i++){
        fscanf(fptr, "%d,", &token);
        array[i] = token;
    }

    fclose(fptr);
    return array;
}

// we want to keep track of how many elements have a 0 in the current bit that is to be masked.
__global__ 
void maskArray(int* result2, int* result, int* array, int mask, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n) {

    // this is the relative index
    result[index] = array[index] & mask;
    result2[index] = !(array[index] & mask);
  } 
}

__global__ 
void prescan(int* indices, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  //extern __shared__ int local_scan[];
  int from = blockIdx.x * blockDim.x;
  int to = blockIdx.x * blockDim.x + blockDim.x; 
   
  for (int d = 1; d < blockDim.x; d *= 2) {
    if (index + 1 - from > d) {
      indices[index] += indices[index-d];
    }
    __syncthreads();
  }
}

__global__
void map(int* result, int from) {
  int index = from + threadIdx.x;
  int to_map = result[from-1];
  result[index] += to_map;
  return;
}

__global__
void copy(int* result, int* array, int* ones, int* zeroes, int n, int pivot, int mask) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n) {
      if (array[index] & mask) {
        int idx = ones[index];
        result[idx + pivot] = array[index];
      }
      else {
        int idx = zeroes[index];
        result[idx] = array[index];
      }
  }
}

void radixSort(int* array, int n) {
    int threads = 1024;
    int blocks = (n + (threads-1)) / threads;
    
    // stores whether each element in array is odd or not (denoted 1 or 0)
    int* ones;
    int* zeroes;
    int* result_copy;  // stores prefix sum of each element
    int* result;  // stores final result, sizeof prefix[n-1]
    int local_array_bytes = sizeof(int)*threads;

    hipMallocManaged(&result_copy, sizeof(int) * n);
    hipMallocManaged(&result, sizeof(int) * n);
    hipMallocManaged(&ones, sizeof(int) * n);
    hipMallocManaged(&zeroes, sizeof(int) * n);

    for (int i = 1; i <= 1024; i <<= 1) {
        maskArray<<<blocks, threads>>>(zeroes, ones, array, i, n);

        hipDeviceSynchronize();

        prescan<<<blocks, threads, local_array_bytes>>>(zeroes, n); 

        for(int i = threads; i < n; i+=threads) {
        map<<<1, threads>>>(zeroes, i); //map last value of previous group of 1024 onto next group of 1024
        hipDeviceSynchronize();
        }

        prescan<<<blocks, threads, local_array_bytes>>>(ones, n); 

        for(int i = threads; i < n; i+=threads) {
        map<<<1, threads>>>(ones, i); //map last value of previous group of 1024 onto next group of 1024
        hipDeviceSynchronize();
        }

        int pivot = zeroes[n-1];

        // so far we've only calculated the positions of elements with 0 in the bit of interest
        // we need to use new index of the last element with a 0 in the bit of interest as an offset 
        // to calculate the positions of elements with 1 in the bit of interest
        // also I think we need to copy results -> input array at the end of each iteration for each bit of interest
  
        hipDeviceSynchronize();
        copy<<<blocks, threads>>>(result, array, ones, zeroes, n, pivot, mask);
        for(int i = 0; i < 10; i++) {
          printf("result[%d]: %d\n", i, result[i]);
        }
    }
  
  }


int main(int argc, char* argv[]){
    int n;
    int* array = fileToArray("inp.txt", &n);
    radixSort(array, n);
    hipFree(array);
  }