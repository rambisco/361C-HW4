
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>

#define NUM_THREADS_A 32
#define NUM_BLOCKS_A 2
#define NUM_THREADS_B 32
#define NUM_BLOCKS_B 2

int* fileToArray(char file1[], int* n){
  FILE* fptr = fopen(file1, "r");
  char* str = (char*) malloc(sizeof(char)*2048);
  int token;
  fscanf(fptr, "%d,", n);
  int* array;
  //int* array = malloc(sizeof(int)*(*n));
  hipMallocManaged(&array, sizeof(int)*(*n)); 
  for(int i = 0; i < *n; i++){
    fscanf(fptr, "%d,", &token);
    array[i] = token;
  }
 fclose(fptr);
 return array;
}

__global__
void sharedBucket(int* array, int* result, int n) {
  __shared__ int local_array[10];  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {

    int numHundreds = array[i] / 100;
    atomicAdd((local_array+numHundreds), 1);
  }
  __syncthreads();
  if ((threadIdx.x | threadIdx.y | threadIdx.z) == 0) {

    for (int i = 0; i < 10; i++) {
      atomicAdd((result+i), local_array[i]);
//      result[i] = local_array[i];
    }
  }
  __syncthreads();
}

void computeSharedBucket(int* array, int n) {

  int* result;
  hipMallocManaged(&result, sizeof(int)*(10));

  for (int i = 0; i < 10; i++) {
    result[i] = 0;
  }

  sharedBucket<<<NUM_BLOCKS_A, NUM_THREADS_A>>>(array, result, n);

  hipDeviceSynchronize();
  for (int i = 0; i < 10; i++) {
    printf("result[%d]: %d\n", i, result[i]);
  }
}

__global__
void bucket(int* array, int* result, int n) {
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {

    int numHundreds = array[i] / 100;
    atomicAdd((result+numHundreds), 1);
  }
}

void computeBucket(int* array, int n) {

  int* result;
  hipMallocManaged(&result, sizeof(int)*(10));

  for (int i = 0; i < 10; i++) {
    result[i] = 0;
  }

  bucket<<<NUM_BLOCKS_B, NUM_THREADS_B>>>(array, result, n);

  hipDeviceSynchronize();
  for (int i = 0; i < 10; i++) {
    printf("result[%d]: %d\n", i, result[i]);
  }
}

int main(int argc, char* argv[]){
  int n;
  int* array = fileToArray("inp.txt", &n);
  /*for(int i = 0; i < 10; i++){
    printf("%d\n", array[i]);
  }*/
  computeBucket(array, n);
  computeSharedBucket(array, n);
  //int min = computeMin(array, n);
  //printf("min: %d\n", min);
  hipFree(array);
}
