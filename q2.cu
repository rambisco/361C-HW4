
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>

#define OUTPUT_FILE_NAME_A "q2a.txt"
#define OUTPUT_FILE_NAME_B "q2b.txt"
#define OUTPUT_FILE_NAME_C "q2c.txt"
#define NUM_THREADS_A 32
#define NUM_BLOCKS_A 2
#define NUM_THREADS_B 32
#define NUM_BLOCKS_B 2

// int* fileToArray(char file1[], int* n){
//   FILE* fptr = fopen(file1, "r");
//   char* str = (char*) malloc(sizeof(char)*2048);
//   int token;
//   fscanf(fptr, "%d,", n);
//   int* array;
//   //int* array = malloc(sizeof(int)*(*n));
//   cudaMallocManaged(&array, sizeof(int)*(*n)); 
//   for(int i = 0; i < *n; i++){
//     fscanf(fptr, "%d,", &token);
//     array[i] = token;
//   }
//  fclose(fptr);
//  return array;
// }

int* fileToArray(char file1[], int* n){
  FILE* fptr = fopen(file1, "r");
  char* str = (char*) malloc(sizeof(char)*2048);
  int token;
  int count = 0;
  while (fscanf(fptr, "%d, ", &token) != EOF) {
    //printf("%dth token: %d\n", count, token);
    count++;
  }
  *n = count;
  //printf("total number of elements: %d\n", *n);
  int* array;
  hipMallocManaged(&array, sizeof(int)*(*n));
  rewind(fptr);
  for(int i = 0; i < *n; i++){
      fscanf(fptr, "%d, ", &token);
      array[i] = token;
  }

  fclose(fptr);
  return array;
}

__global__
void sharedBucket(int* array, int* result, int n) {
  __shared__ int local_array[10];  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {

    int numHundreds = array[i] / 100;
    atomicAdd((local_array+numHundreds), 1);
  }
  __syncthreads();
  if ((threadIdx.x | threadIdx.y | threadIdx.z) == 0) {

    for (int i = 0; i < 10; i++) {
      atomicAdd((result+i), local_array[i]);
//      result[i] = local_array[i];
    }
  }
  __syncthreads();
}

void computeSharedBucket(int* array, int n) {

  int* result;
  hipMallocManaged(&result, sizeof(int)*(10));

  for (int i = 0; i < 10; i++) {
    result[i] = 0;
  }

  sharedBucket<<<NUM_BLOCKS_A, NUM_THREADS_A>>>(array, result, n);

  hipDeviceSynchronize();
  FILE *output = fopen(OUTPUT_FILE_NAME_B, "w");
  if(output == NULL) printf("failed to open file %s\n", OUTPUT_FILE_NAME_B);
  fprintf(output, "%d", result[0]);
  for(int i = 1; i < 10 ; i++) {
    fprintf(output, ", %d", result[i]);
  }
  fclose(output);  
}

__global__
void bucket(int* array, int* result, int n) {
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {

    int numHundreds = array[i] / 100;
    atomicAdd((result+numHundreds), 1);
  }
}

__global__ 
void prescan(int* array, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  extern __shared__ int local_scan[];
   
  for (int d = 1; d < 10; d *= 2) {
    if (index + 1 > d && index < 10) {
      array[index] += array[index-d];
    }
    __syncthreads();
  }
}

void computeBucket(int* array, int n) {

  int* result;
  hipMallocManaged(&result, sizeof(int)*(10));
  for (int i = 0; i < 10; i++) {
    result[i] = 0;
  }

  bucket<<<NUM_BLOCKS_B, NUM_THREADS_B>>>(array, result, n);

  hipDeviceSynchronize();

  FILE *output = fopen(OUTPUT_FILE_NAME_A, "w");
  if(output == NULL) printf("failed to open file %s\n", OUTPUT_FILE_NAME_A);
  fprintf(output, "%d", result[0]);
  for(int i = 1; i < 10 ; i++) {
    fprintf(output, ", %d", result[i]);
  }
  fclose(output);  

  prescan<<<1, 10>>>(result, 10); 

  
  output = fopen(OUTPUT_FILE_NAME_C, "w");
  if(output == NULL) printf("failed to open file %s\n", OUTPUT_FILE_NAME_C);
  fprintf(output, "%d", result[0]);
  for(int i = 1; i < 10 ; i++) {
    fprintf(output, ", %d", result[i]);
  }
  fclose(output);  
}

int main(int argc, char* argv[]){
  int n;
  int* array = fileToArray("inp.txt", &n);
  /*for(int i = 0; i < 10; i++){
    printf("%d\n", array[i]);
  }*/
  computeBucket(array, n);
  computeSharedBucket(array, n);
  //int min = computeMin(array, n);
  //printf("min: %d\n", min);
  hipFree(array);
}
