
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>

#define NUM_THREADS 32
#define NUM_BLOCKS 2

int* fileToArray(char file1[], int* n){
  FILE* fptr = fopen(file1, "r");
  char* str = (char*) malloc(sizeof(char)*2048);
  int token;
  fscanf(fptr, "%d,", n);
  int* array;
  //int* array = malloc(sizeof(int)*(*n));
  hipMallocManaged(&array, sizeof(int)*(*n)); 
  for(int i = 0; i < *n; i++){
    fscanf(fptr, "%d,", &token);
    array[i] = token;
  }
 fclose(fptr);
 return array;
}

__global__
void countOdds(int* array, int* result, int n) {
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    atomicAdd(result, 1);
  }
}

__global__
void odds(int* array, int* result, int n){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    atomicAdd(result, 1);
  }


}

void copyOdds(int* array, int n) {

  int* result;
  hipMallocManaged(&result, sizeof(int));


  countOdds<<<NUM_BLOCKS, NUM_THREADS>>>(array, result, n);
  hipDeviceSynchronize();




}

int main(int argc, char* argv[]){
  int n;
  int* array = fileToArray("inp.txt", &n);
  copyOdds(array, n);
  //int min = computeMin(array, n);
  //printf("min: %d\n", min);
  hipFree(array);
}