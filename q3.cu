
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>

#define NUM_THREADS 512
#define NUM_BLOCKS 1
#define ZERO_BANK_CONFLICTS 1
#define OUTPUT_FILE_NAME "q3.txt"
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) \
 ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif

// int* fileToArray(char file1[], int* n){
//   FILE* fptr = fopen(file1, "r");
//   char* str = (char*) malloc(sizeof(char)*2048);
//   int token;
//   fscanf(fptr, "%d,", n);
//   int* array;
//   //int* array = malloc(sizeof(int)*(*n));
//   cudaMallocManaged(&array, sizeof(int)*(*n)); 
//   for(int i = 0; i < *n; i++){
//     fscanf(fptr, "%d,", &token);
//     array[i] = token;
//   }
//  fclose(fptr);
//  return array;
// }

int* fileToArray(char file1[], int* n){
  FILE* fptr = fopen(file1, "r");
  //char* str = (char*) malloc(sizeof(char)*2048);
  int token;
  int count = 0;
  while (fscanf(fptr, "%d, ", &token) != EOF) {
    //("%dth token: %d\n", count, token);
    count++;
  }
  *n = count;
  //printf("total number of elements: %d\n", *n);
  int* array;
  hipMallocManaged(&array, sizeof(int)*(*n));
  rewind(fptr);
  for(int i = 0; i < *n; i++){
      fscanf(fptr, "%d, ", &token);
      array[i] = token;
  }

  fclose(fptr);
  return array;
}

__global__ 
void odds(int* result, int* array, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n) {
    result[index] = array[index] % 2;
  } 
}

__global__ 
void prescan(int* result, int* odds, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  extern __shared__ int local_scan[];
  int from = blockIdx.x * blockDim.x;
  int to = blockIdx.x * blockDim.x + blockDim.x; 
   
  for (int d = 1; d < blockDim.x; d *= 2) {
    if (index + 1 - from > d) {
      odds[index] += odds[index-d];
    }
    __syncthreads();
  }
  result[index] = odds[index];
}

__global__
void map(int* result, int from) {
  int index = from + threadIdx.x;
  int to_map = result[from-1];
  result[index] += to_map;
  return;
}

__global__
void copy(int* result, int* odds, int* array, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n) {
      if (array[index] % 2 == 1) {
        int idx = odds[index];
        result[idx] = array[index];
      }
  }
}

void copyOdds(int* array, int n) {
  int threads = 1024;
  int blocks = (n + (threads-1)) / threads;
  

  int* ones;  // stores whether each element in array is odd or not (denoted 1 or 0)
  int* prefix;  // stores prefix sum of each element
  int* result;  // stores final result, sizeof prefix[n-1]
  int local_array_bytes = sizeof(int)*threads;

  hipMallocManaged(&ones, sizeof(int) * n);
  hipMallocManaged(&prefix, sizeof(int) * n);

  odds<<<blocks, threads>>>(ones, array, n);
  printf("ones[999999]: %d\n", ones[999999]);

  hipDeviceSynchronize();

  prescan<<<blocks, threads, local_array_bytes>>>(prefix, ones, n); 


  hipDeviceSynchronize();

  for(int i = threads; i < n; i+=threads) {
    map<<<1, threads>>>(prefix, i); //map last value of previous group of 1024 onto next group of 1024
    hipDeviceSynchronize();
  }

  printf("prescan[999998]: %d, prescan[999999]: %d\n", prefix[999998], prefix[999999]);

  int maxOdds = prefix[n] + 1;
  printf("max number of odds: %d\n", prefix[n]);

  hipMallocManaged(&result, sizeof(int) * maxOdds);

  copy<<<blocks, threads>>>(result, prefix, array, n);

  hipDeviceSynchronize();

   for(int i = maxOdds - 10; i < maxOdds; i++) {
     printf("index: %d result: %d\n", i, result[i]);
   }

  // FILE *output = fopen(OUTPUT_FILE_NAME, "w");
  // if(output == NULL) printf("failed to open file %s\n", OUTPUT_FILE_NAME);
  // fprintf(output, "%d", result[0]);
  // for(int i = 0; i < maxOdds ; i++) {
  //   fprintf(output, ",%d", result[i]);
  // }
  // fclose(output);
}

int main(int argc, char* argv[]){
  int n;
  int* array = fileToArray("inp.txt", &n);
  
  // for (int i = 0; i < n; i++) {
  //   printf("Is %d odds? %d\n", array[i], ((array[i] % 2) == 0) ? 0 : 1);
  // }
  copyOdds(array, n);
  //int min = computeMin(array, n);
  //printf("min: %d\n", min);
  hipFree(array);
}