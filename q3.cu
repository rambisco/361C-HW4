
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>

#define NUM_THREADS 512
#define NUM_BLOCKS 1
#define ZERO_BANK_CONFLICTS 1
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) \
 ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif

int* fileToArray(char file1[], int* n){
  FILE* fptr = fopen(file1, "r");
  char* str = (char*) malloc(sizeof(char)*2048);
  int token;
  fscanf(fptr, "%d,", n);
  int* array;
  //int* array = malloc(sizeof(int)*(*n));
  hipMallocManaged(&array, sizeof(int)*(*n)); 
  for(int i = 0; i < *n; i++){
    fscanf(fptr, "%d,", &token);
    array[i] = token;
  }
 fclose(fptr);
 return array;
}

// __global__
// void countOdds(int* array, int* result, int n) {
  
//   int index = blockIdx.x * blockDim.x + threadIdx.x;
//   int stride = blockDim.x * gridDim.x;
//   for (int i = index; i < n; i += stride) {
//     atomicAdd(result, 1);
//   }
// }

// __global__
// void odds(int* array, int* result, int n){
//   int index = blockIdx.x * blockDim.x + threadIdx.x;
//   int stride = blockDim.x * gridDim.x;
//   for (int i = index; i < n; i += stride) {
//     atomicAdd(result, 1);
//   }


// }

__global__ void prescan(int* result, int* array, int n) {
  n = NUM_THREADS; //we cant do more than this yet
  extern __shared__ int counts[];
  int thid = threadIdx.x;
  int offset = 1;

  int ai = thid;
  int bi = thid + (n/2);
  int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
  int bankOffsetB = CONFLICT_FREE_OFFSET(ai);

  counts[ai + bankOffsetA] = ((array[ai] % 2) == 0) ? 0 : 1;
  counts[bi + bankOffsetB] = ((array[bi] % 2) == 0) ? 0 : 1;


  for (int d = n>>1; d > 0; d >>= 1) {
    __syncthreads();
    if (thid < d) {
      int ai = offset*(2*thid+1)-1;
      int bi = offset*(2*thid+2)-1;
      ai += CONFLICT_FREE_OFFSET(ai);
      bi += CONFLICT_FREE_OFFSET(bi); 
      counts[bi] += counts[ai]; 
    }
    offset *= 2;
  }
  
  if (thid == 0) {
    counts[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
  }

  for (int d = 1; d < n; d *= 2) {
    offset >>= 1;
    __syncthreads();
    if (thid < d) {
      int ai = offset*(2*thid+1)-1;
      int bi = offset*(2*thid+2)-1;
      ai += CONFLICT_FREE_OFFSET(ai);
      bi += CONFLICT_FREE_OFFSET(bi); 
      int t = counts[ai];
      counts[ai] = counts[bi];
      counts[bi] += t;
    }
  }
  __syncthreads();
  result[ai] = counts[ai + bankOffsetA];
  result[bi] = counts[bi + bankOffsetB]; 
}

void copyOdds(int* array, int n) {

  int* result;
  hipMallocManaged(&result, sizeof(int));


  prescan<<<NUM_BLOCKS, NUM_THREADS>>>(result, array, n);
  hipDeviceSynchronize();
  for(int i = 0; i < NUM_THREADS; i++) {
    printf("number of odds before index %d is: %d\n", i, result[i]);
  }

}

int main(int argc, char* argv[]){
  int n;
  int* array = fileToArray("inp.txt", &n);
  copyOdds(array, n);
  //int min = computeMin(array, n);
  //printf("min: %d\n", min);
  hipFree(array);
}