
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>

#define NUM_THREADS 512
#define NUM_BLOCKS 1
#define ZERO_BANK_CONFLICTS 1
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) \
 ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif

int* fileToArray(char file1[], int* n){
  FILE* fptr = fopen(file1, "r");
  char* str = (char*) malloc(sizeof(char)*2048);
  int token;
  fscanf(fptr, "%d,", n);
  int* array;
  //int* array = malloc(sizeof(int)*(*n));
  hipMallocManaged(&array, sizeof(int)*(*n)); 
  for(int i = 0; i < *n; i++){
    fscanf(fptr, "%d,", &token);
    array[i] = token;
  }
 fclose(fptr);
 return array;
}

// __global__
// void countodds(int* array, int* result, int n) {
  
//   int index = blockIdx.x * blockDim.x + threadIdx.x;
//   int stride = blockDim.x * gridDim.x;
//   for (int i = index; i < n; i += stride) {
//     atomicAdd(result, 1);
//   }
// }

// __global__
// void odds(int* array, int* result, int n){
//   int index = blockIdx.x * blockDim.x + threadIdx.x;
//   int stride = blockDim.x * gridDim.x;
//   for (int i = index; i < n; i += stride) {
//     atomicAdd(result, 1);
//   }


// }

// __global__ void prescan(int* result, int* array, int n) {
  //n = NUM_THREADS; //we cant do more than this yet
  // extern __shared__ int counts[];
  // int thid = blockIdx.x * blockDim.x + threadIdx.x;
  //int thid = threadIdx.x;
  // int offset = 1;

  // int ai = thid;
  // int bi = thid + (n/2);
  // int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
  // int bankOffsetB = CONFLICT_FREE_OFFSET(ai);

  // counts[ai + bankOffsetA] = ((array[ai] % 2) == 0) ? 0 : 1;
  // counts[bi + bankOffsetB] = ((array[bi] % 2) == 0) ? 0 : 1;

//   for (int d = n>>1; d > 0; d >>= 1) {
//     __syncthreads();
//     if (thid < d) {
//       int ai = offset*(2*thid+1)-1;
//       int bi = offset*(2*thid+2)-1;
//       ai += CONFLICT_FREE_OFFSET(ai);
//       bi += CONFLICT_FREE_OFFSET(bi); 
//       counts[bi] += counts[ai]; 
//     }
//     offset *= 2;
//   }
  
//   if (thid == 0) {
//     counts[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
//   }

//   for (int d = 1; d < n; d *= 2) {
//     offset >>= 1;
//     __syncthreads();
//     if (thid < d) {
//       int ai = offset*(2*thid+1)-1;
//       int bi = offset*(2*thid+2)-1;
//       ai += CONFLICT_FREE_OFFSET(ai);
//       bi += CONFLICT_FREE_OFFSET(bi); 
//       int t = counts[ai];
//       counts[ai] = counts[bi];
//       counts[bi] += t;
//     }
//   }
//   __syncthreads();
//   result[ai] = counts[ai + bankOffsetA];
//   result[bi] = counts[bi + bankOffsetB]; 
// }
__global__ 
void odds(int* result, int* array, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n) {
    result[index] = array[index] % 2;
  } 
}

__global__ 
void prescan(int* result, int* odds, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  for (int d = 1; d < n; d *= 2) {
    if (index+1 > d && index < n) {
      odds[index] += odds[index-d];
    }
    __syncthreads();
  }
  result[index] = odds[index];
}

__global__
void copy(int* result, int* odds, int* array, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  // int stride = blockDim.x * gridDim.x;
  // for (int i = index; i < n; i += stride) {
  //   if (array[i] % 2 == 1) {
  //     int idx = odds[i];
  //     result[idx] = array[i];
  //   }
  // }
  if (index < n) {
      if (array[index] % 2 == 1) {
        int idx = odds[index];
        result[idx] = array[index];
      }
  }
}
 

//   printf("max number of odds: %d\n", odds[n-1]);

void copyOdds(int* array, int n) {
  int threads = 1024;
  int blocks = (n + (threads-1)) / threads;
  

  int* ones;  // stores whether each element in array is odd or not (denoted 1 or 0)
  int* prefix;  // stores prefix sum of each element
  int* result;  // stores final result, sizeof prefix[n-1]

  hipMallocManaged(&ones, sizeof(int) * n);
  hipMallocManaged(&prefix, sizeof(int) * n);

  odds<<<blocks, threads>>>(ones, array, n);

  hipDeviceSynchronize();

  prescan<<<blocks, threads>>>(prefix, ones, n);

  hipDeviceSynchronize();

  int maxOdds = prefix[n-1];
  printf("max number of odds: %d\n", prefix[n-1]);

  for(int i = 0; i < threads; i++) {
    printf("is index %d odds? %d\n", i, prefix[i]);
  }

  hipMallocManaged(&result, sizeof(int) * maxOdds);

  copy<<<blocks, threads>>>(result, prefix, array, n);

  hipDeviceSynchronize();

  // for(int i = 1; i < threads; i++) {
  //   printf("index: %d result: %d\n", i, result[i]);
  // }

}

int main(int argc, char* argv[]){
  int n;
  int* array = fileToArray("inp.txt", &n);
  
  // for (int i = 0; i < n; i++) {
  //   printf("Is %d odds? %d\n", array[i], ((array[i] % 2) == 0) ? 0 : 1);
  // }
  copyOdds(array, n);
  //int min = computeMin(array, n);
  //printf("min: %d\n", min);
  hipFree(array);
}