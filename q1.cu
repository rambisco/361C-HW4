
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>


int* fileToArray(char file1[], int* n){
  FILE* fptr = fopen(file1, "r");
  char* str = (char*) malloc(sizeof(char)*2048);
  int token;
  fscanf(fptr, "%d,", n);
  int* array;
  //int* array = malloc(sizeof(int)*(*n));
  hipMallocManaged(&array, sizeof(int)*(*n)); 
  for(int i = 0; i < *n; i++){
    fscanf(fptr, "%d,", &token);
    array[i] = token;
  }
 fclose(fptr);
 return array;
}

__global__
void min(int* array, int n){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int currentMin = INT_MAX;
  for(int i = index; i < n; i += stride){
    if(array[i] < currentMin){
      currentMin = array[i];
    }
  }
  array[index] = currentMin;
}

int computeMin(int* array, int n){

  int numThreads = 1;
  int numBlocks = 1;
  min<<<numBlocks, numThreads>>>(array, n);

  hipDeviceSynchronize();

  int minNum = INT_MAX;
  for(int i = 0; i < numThreads; i++){
    if(array[i] < minNum){
      minNum = array[i];
    }
  }
  return minNum;
}


int main(int argc, char* argv[]){
  int n;
  int* array = fileToArray("inp.txt", &n);
  for(int i = 0; i < 10; i++){
    printf("%d\n", array[i]);
  }
  int min = computeMin(array, n);
  printf("min: %d\n", min);
  hipFree(array);
}
