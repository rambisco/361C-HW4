
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>


#define OUTPUT_FILE_NAME_B "q1b.txt"
#define OUTPUT_FILE_NAME_MIN "q1a.txt"
#define NUM_THREADS_A 32
#define NUM_BLOCKS_A 2
#define NUM_THREADS_B 32
#define NUM_BLOCKS_B 2
// int* fileToArray(char file1[], int* n){
//   FILE* fptr = fopen(file1, "r");
//   char* str = (char*) malloc(sizeof(char)*2048);
//   int token;
//   fscanf(fptr, "%d,", n);
//   int* array;
//   //int* array = malloc(sizeof(int)*(*n));
//   cudaMallocManaged(&array, sizeof(int)*(*n)); 
//   for(int i = 0; i < *n; i++){
//     fscanf(fptr, "%d,", &token);
//     array[i] = token;
//   }
//  fclose(fptr);
//  return array;
// }

int* fileToArray(char file1[], int* n){
  FILE* fptr = fopen(file1, "r");
  char* str = (char*) malloc(sizeof(char)*2048);
  int token;
  int count = 0;
  while (fscanf(fptr, "%d, ", &token) != EOF) {
    //("%dth token: %d\n", count, token);
    count++;
  }
  *n = count;
  //printf("total number of elements: %d\n", *n);
  int* array;
  hipMallocManaged(&array, sizeof(int)*(*n));
  rewind(fptr);
  for(int i = 0; i < *n; i++){
      fscanf(fptr, "%d, ", &token);
      array[i] = token;
  }

  fclose(fptr);
  return array;
}
__global__
void lastDigit(int* array, int* result, int n) {
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    result[i] = array[i] % 10;
  }
}

__global__
void min(int* array, int n){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int currentMin = INT_MAX;
  for(int i = index; i < n; i += stride){
    if(array[i] < currentMin){
      currentMin = array[i];
    }
  }
  array[index] = currentMin;
}

int computeMin(int* array, int n){

  min<<<NUM_BLOCKS_A, NUM_THREADS_A>>>(array, n);

  hipDeviceSynchronize();

  int minNum = INT_MAX;
  for(int i = 0; i < NUM_THREADS_A; i++){
    if(array[i] < minNum){
      minNum = array[i];
    }
  }
  return minNum;
}

void computeLastDigit(int* array, int n) {
  int* result;
  hipMallocManaged(&result, sizeof(int)*(n));
  lastDigit<<<NUM_BLOCKS_B, NUM_THREADS_B>>>(array, result, n);

  hipDeviceSynchronize();
  // for (int i = 0; i < 10; i++) {
  //   printf("array[%d]: %d, result[%d]: %d\n", i, array[i], i, result[i]);
  // }
  FILE *output = fopen(OUTPUT_FILE_NAME_B, "w");
  if(output == NULL) printf("failed to open file %s\n", OUTPUT_FILE_NAME_B);
  fprintf(output, "%d", result[0]);
  for(int i = 1; i < n ; i++) {
    fprintf(output, ", %d", result[i]);
  }
  fclose(output);
}

int main(int argc, char* argv[]){
  int n;
  int* array = fileToArray("inp.txt", &n);
  //printf("Number of elements in array: %d\n", n);
  // for (int i = 0; i < n; i++) {
  //   printf("%d, ", array[i]);
  // }
  /*for(int i = 0; i < 10; i++){
    printf("%d\n", array[i]);
  }*/
  computeLastDigit(array, n);
  int min = computeMin(array, n);
  
  FILE *output = fopen(OUTPUT_FILE_NAME_MIN, "w");
  if(output == NULL) printf("failed to open file %s\n", OUTPUT_FILE_NAME_MIN);
  fprintf(output, "%d", min);
  fclose(output);
  hipFree(array);
}
