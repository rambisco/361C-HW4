
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>

#define NUM_THREADS_A 32
#define NUM_BLOCKS_A 2
#define NUM_THREADS_B 32
#define NUM_BLOCKS_B 2
int* fileToArray(char file1[], int* n){
  FILE* fptr = fopen(file1, "r");
  char* str = (char*) malloc(sizeof(char)*2048);
  int token;
  fscanf(fptr, "%d,", n);
  int* array;
  //int* array = malloc(sizeof(int)*(*n));
  hipMallocManaged(&array, sizeof(int)*(*n)); 
  for(int i = 0; i < *n; i++){
    fscanf(fptr, "%d,", &token);
    array[i] = token;
  }
 fclose(fptr);
 return array;
}
__global__
void lastDigit(int* array, int* result, int n) {
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    result[i] = array[i] % 10;
  }
}

__global__
void min(int* array, int n){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int currentMin = INT_MAX;
  for(int i = index; i < n; i += stride){
    if(array[i] < currentMin){
      currentMin = array[i];
    }
  }
  array[index] = currentMin;
}

int computeMin(int* array, int n){

  min<<<NUM_BLOCKS_A, NUM_THREADS_A>>>(array, n);

  hipDeviceSynchronize();

  int minNum = INT_MAX;
  for(int i = 0; i < NUM_THREADS_A; i++){
    if(array[i] < minNum){
      minNum = array[i];
    }
  }
  return minNum;
}

void computeLastDigit(int* array, int n) {
  int* result;
  hipMallocManaged(&result, sizeof(int)*(n));
  lastDigit<<<NUM_BLOCKS_B, NUM_THREADS_B>>>(array, result, n);

  hipDeviceSynchronize();
  for (int i = 0; i < 10; i++) {
    printf("array[%d]: %d, result[%d]: %d\n", i, array[i], i, result[i]);
  }
}

int main(int argc, char* argv[]){
  int n;
  int* array = fileToArray("inp.txt", &n);
  /*for(int i = 0; i < 10; i++){
    printf("%d\n", array[i]);
  }*/
  computeLastDigit(array, n);
  int min = computeMin(array, n);
  printf("min: %d\n", min);
  hipFree(array);
}
